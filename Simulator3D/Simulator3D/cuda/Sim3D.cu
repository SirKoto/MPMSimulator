#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include <vector>
#include "Simulator_3D_GPU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

extern "C"
void loadParticles2GPU(const S3::Particle * particles, size_t num);
extern "C"
void loadGrid2GPU(size_t num);
extern "C"
void loadPhysicsGrid2GPU(const Eigen::Array3f * grid, size_t num);
extern "C"
void loadProperties2GPU(const S3::property * properties, size_t num);
extern "C"
void storeParticles2Array(S3::Particle * particles, size_t num);

__device__ S3::Particle* d_particles;
__device__ Eigen::Array4f* d_grid;
__device__ __constant__ Eigen::Array3f* d_physicsGrid;
__device__ __constant__ S3::property* d_properties;

#define GRID_SIZE 128.0f
#define D_SIZE = 1.0f/128.0f
#define G = Eigen::Array3f(0.0f, -10.0f, 0.0f);
#define getInd(x, y, z) (((((x) << 7) | (y)) << 7) | (z))

void loadParticles2GPU(const S3::Particle* particles, size_t num)
{
	S3::Particle* tmp;
	gpuErrchk(hipMalloc((S3::Particle**)&tmp, num * sizeof(S3::Particle)));
	gpuErrchk(hipMemcpy(tmp, particles, num * sizeof(S3::Particle), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_particles), (void**)&tmp, sizeof(S3::Particle*)));
}

void loadGrid2GPU(size_t num)
{
	assert(static_cast<size_t>(GRID_SIZE) == num);

	num = num * num * num;
	Eigen::Array4f* tmp;
	gpuErrchk(hipMalloc((Eigen::Array4f**)&tmp, num * sizeof(Eigen::Array4f)));
	gpuErrchk(hipMemset((Eigen::Array4f**) & tmp, 0, num * sizeof(Eigen::Array4f)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_grid), (void**)&tmp, sizeof(Eigen::Array4f*)));
}

void loadPhysicsGrid2GPU(const Eigen::Array3f* grid, size_t num)
{
	assert(static_cast<size_t>(GRID_SIZE) == num);

	num = num * num * num;
	Eigen::Array3f* tmp;
	gpuErrchk(hipMalloc((Eigen::Array3f**) & tmp, num * sizeof(Eigen::Array3f)));
	gpuErrchk(hipMemcpy(tmp, grid, num * sizeof(Eigen::Array3f), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_physicsGrid), (void**)&tmp, sizeof(Eigen::Array3f*)));
}

void loadProperties2GPU(const S3::property* properties, size_t num)
{
	S3::property* tmp;
	gpuErrchk(hipMalloc((S3::property**) & tmp, num * sizeof(S3::property)));
	gpuErrchk(hipMemcpy(tmp, properties, num * sizeof(S3::property), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_properties), (void**)&tmp, sizeof(S3::property*)));
}

void storeParticles2Array(S3::Particle* particles, size_t num)
{
	S3::Particle* tmp;
	gpuErrchk(hipMemcpyFromSymbol(&tmp,HIP_SYMBOL(d_particles), num * sizeof(S3::Particle)));
	gpuErrchk(hipMemcpy(&particles, tmp, num * sizeof(S3::Particle), hipMemcpyDeviceToHost));
}