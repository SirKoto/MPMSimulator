#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdint>
#include <stdio.h>
#include <vector>
#include "Simulator_3D_GPU.h"
#define GENERATE_IMPLEMENTATION_MM
#include "MinMath.h"

#include "svd3_cuda.cuh"

using namespace mm;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

struct ParticleGPU
{
	vec3 pos, v; // position and velocity

	mat3 F, C; // Deformation gradient and C from APIC

	float Jp; // Determinat of plastic F (Jacobian).

	const int prop_id; // material id
};

extern "C"
void loadParticles2GPU(const S3::Particle * particles, size_t num);
extern "C"
void loadGrid2GPU(size_t num);
extern "C"
void loadPhysicsGrid2GPU(const Eigen::Array3f * grid, size_t num);
extern "C"
void loadProperties2GPU(const S3::property * properties, size_t num);
extern "C"
void storeParticles2Array(S3::Particle * particles, size_t num);
extern "C"
void runIterationGPU(float dt);

__device__ ParticleGPU* d_particles;
__device__ __constant__ unsigned int d_numParticles;
unsigned int h_numParticles;
__device__ vec4* d_grid;
vec4* h_grid;
__device__ __constant__ vec3* d_physicsGrid;
__device__ __constant__ S3::property* d_properties;

#define GRID_SIZE 128.0f
#define GRID_SIZE_I 128
#define D_SIZE (1.0f/128.0f)
#define G vec3{0.0f, -10.0f, 0.0f}
#define getInd(x, y, z) (((((x) << 7) | (y)) << 7) | (z))

#define BLOCK_SIZE 32

__global__ void P2G_NeoHookean(const float dt) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

if (idx < d_numParticles) {
	const ParticleGPU& p = d_particles[idx];
	// Discretize position 
	const vec3 cell_if = mm::mul(p.pos, GRID_SIZE);
	int cell_x = cell_if.x, cell_y = cell_if.y, cell_z = cell_if.z; // floor
	// Vector from cell center -> particle
	vec3 distFromCenter = mm::sub(cell_if, mm::floor(cell_if));
	mm::add_to(&distFromCenter, -0.5f); // center at point 0,0,0


	// Interpolation function N matrix
	vec3 weights[3];
	{
		{
			vec3 center = mm::sub(0.5f, distFromCenter);
			mm::square_in(&center); // Squared
			weights[0] = mm::mul(center, 0.5f);
		}
		{
			vec3 center = distFromCenter;
			mm::square_in(&center); // Squared
			weights[1] = mm::sub(0.75f, center);
		}
		{
			vec3 center = mm::add(distFromCenter, 0.5f);
			mm::square_in(&center); // Squared
			weights[2] = mm::mul(center, 0.5f);
		}
	}

	// Load material properies
	const S3::property& p_prop = d_properties[p.prop_id];

	// Lame parameters
	const float e = expf(p_prop.hardening * (1.0f - p.Jp));
	const float mu = p_prop.mu * e;
	const float lambda = p_prop.lambda * e;

	const float mass = p_prop.mass;

	const float J = mm::determinant(p.F);

	// ----------  AFFINE MATRIX ---------- //
	mat3 affine = mm::mul(p.C, mass);
	if(true){
		mat3 PF_t = mm::mul_trans(p.F);
		mm::add_to(&PF_t, -1.0f); // Add only to diagonal
		mm::mul_in(&PF_t, mu); // Multiply by mu
		mm::add_to(&PF_t, lambda * logf(J));

		const float Dinv = (4.0f * GRID_SIZE * GRID_SIZE);
		const float K = -dt * p_prop.volume * Dinv;
		mm::mul_in(&PF_t, K); // Compute stress
		//const Eigen::Matrix3f stress = (-dt * p_prop.volume * Dinv) * PF_t;
		mm::add_to(&affine, PF_t);
	}

	// ----------  PARTICLE TRANSFERENCE ---------- //
	// This can be improved by using the previously computed distFromCenter
	vec3 cell_dist0 = mm::add(mm::floor(cell_if), -1.0f);
	cell_dist0 = mm::sub(cell_dist0, cell_if);
	mm::add_to(&cell_dist0, 0.5f);

	vec3 moment = mm::mul(p.v, mass);

	vec4 moment_mass0 = { moment.x, moment.y, moment.z, mass }; // moment and particle mass
	
	vec3 contribution = mm::mul(affine, cell_dist0);
	mm::mul_in(&contribution, D_SIZE);
	mm::add_to(&moment_mass0, contribution); // Add contribution to momentum of the first cell

	const vec3 kstep = mm::mul(mm::col2(affine), D_SIZE);// affine.col(2)* D_SIZE;
	const vec3 jSemiStep = mm::mul(mm::col1(affine), D_SIZE); //(affine.col(1) * D_SIZE).array();
	const vec3 jstep = mm::sub(jSemiStep, mm::mul(kstep, 3.0f));//jSemiStep -(3.0f * kstep);
	const vec3 iSemiStep = mm::mul(mm::col0(affine), D_SIZE);
	const vec3 istep = mm::sub(iSemiStep, mm::mul(jSemiStep, 3.0f)); //(affine.col(0) * D_SIZE).array() - (3.0f * jSemiStep);

	float w;
	unsigned int index;
	for (int i = -1; i < 2; ++i)
	{
		for (int j = -1; j < 2; ++j)
		{
			for (int k = -1; k < 2; ++k)
			{
				w = weights[i + 1].x * weights[j + 1].y * weights[k + 1].z;
				vec4 value = mm::mul(moment_mass0, w);
				index = getInd(cell_x + i, cell_y + j, cell_z + k);
				vec4* ptr = d_grid + index;

				atomicAdd(reinterpret_cast<float*>(&((*ptr).x)), value.x);
				atomicAdd(reinterpret_cast<float*>(&((*ptr).y)), value.y);
				atomicAdd(reinterpret_cast<float*>(&((*ptr).z)), value.z);
				atomicAdd(reinterpret_cast<float*>(&((*ptr).w)), value.w);

				mm::add_to(&moment_mass0, kstep);
			}
			mm::add_to(&moment_mass0, jstep);
		}
		mm::add_to(&moment_mass0, istep);
	}
}
}


__global__ void GridProcessing(const float dt) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int k = idx % GRID_SIZE_I;
	unsigned int j = (idx / GRID_SIZE_I) % GRID_SIZE_I;
	unsigned int i = (idx / (GRID_SIZE_I * GRID_SIZE_I)) % GRID_SIZE_I;

	vec4 cell = d_grid[idx]; // reference


	if (cell.w > 0.0f)
	{
		// ----------  MOMENTUM 2 VELOCITY---------- //
		cell.x /= cell.w;
		cell.y /= cell.w;
		cell.z /= cell.w;

		// Gravity
		mm::add_to(&cell, mm::mul(G, dt));
		// ----------  LIMITS ---------- //
		if (i < 2 && cell.x < 0.0f)
		{
			cell.x = 0.0f;
		}
		else if (i > GRID_SIZE - 3 && cell.x > 0.0f)
		{
			cell.x = 0.0f;
		}


		if (j < 2 && cell.y < 0.0f)
		{
			cell.y = 0.0f;
		}
		else if (j > GRID_SIZE - 3 && cell.y > 0.0f)
		{
			cell.y = 0.0f;
		}

		if (k < 2 && cell.z < 0.0f)
		{
			cell.z = 0.0f;
		}
		else if (k > GRID_SIZE - 3 && cell.z > 0.0f)
		{
			cell.z = 0.0f;
		}

		// ----------  PHYSICS ---------- //
		const vec3& normalPhyisics = d_physicsGrid[idx];
		// velocity dot normal
		float dot = mm::dot3(cell, normalPhyisics);
		// If oposed
		if (dot < 0.0f)
		{
			assert(false);
			// Remove normal velocity
			vec3 r = mm::mul(normalPhyisics, -dot);
			mm::add_to(&cell, r);
		}

		d_grid[idx] = cell;
	}
}




__global__ void G2P(const float dt) {
unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < d_numParticles) {
	ParticleGPU& p = d_particles[idx];
	
	// compute the center 
	const vec3 cell_if = mm::mul(p.pos, GRID_SIZE);
	// Vector from cell center -> particle
	vec3 distFromCenter = mm::sub(cell_if, mm::floor(cell_if));
	mm::add_to(&distFromCenter, -0.5f); // center at point 0,0,0

	// Interpolation function N matrix
	vec3 weights[3];
	{
		{
			vec3 center = mm::sub(0.5f, distFromCenter);
			mm::square_in(&center); // Squared
			weights[0] = mm::mul(center, 0.5f);
		}
		{
			vec3 center = distFromCenter;
			mm::square_in(&center); // Squared
			weights[1] = mm::sub(0.75f, center);
		}
		{
			vec3 center = mm::add(distFromCenter, 0.5f);
			mm::square_in(&center); // Squared
			weights[2] = mm::mul(center, 0.5f);
		}
	}

	// Set to zero velocity and C, befor transfer
	mm::set_zero(&p.C);
	mm::set_zero(&p.v);

	// ----------  GRID TRANSFER ---------- //
	// First cell
	int cell_x = cell_if.x, cell_y = cell_if.y, cell_z = cell_if.z;
	cell_x -= 1; cell_y -= 1; cell_z -= 1;
	// vector particle -> center cell
	vec3 cell_dist = mm::sub(mm::floor(cell_if), cell_if);
	mm::add_to(&cell_dist, -0.5f); // -1 + 0.5
	float w;
	for (int i = -1; i < 2; ++i)
	{
		for (int j = -1; j < 2; ++j)
		{
			for (int k = -1; k < 2; ++k)
			{
				w = weights[i + 1].x * weights[j + 1].y * weights[k + 1].z;
				const vec3 cell_v = d_grid[getInd(cell_x, cell_y, cell_z)];

				vec3 contri = mm::mul(cell_v, w);
				mm::add_to(&p.v, contri);

				mm::add_outter_product(&p.C, contri, cell_dist);

				cell_z += 1;
				cell_dist.z += 1.0f;
			}
			cell_z -= 3;
			cell_y += 1;

			cell_dist.z -= 3.0f;
			cell_dist.y += 1.0f;
		}
		cell_y -= 3;
		cell_x += 1;

		cell_dist.y -= 3.0f;
		cell_dist.x += 1.0f;
	}
	// Apply D^-1 to get C
	mm::mul_in(&p.C, 4.0f * GRID_SIZE);

	// ----------  ADVECTION ---------- //
	//Eigen::Array3f tmp = p.pos;
	mm::add_to(&p.pos, mm::mul(p.v, dt));
	//printf("Pos  %f, %f, %f, %f\n", p.pos.x, p.pos.y, p.pos.z, dt);

	//printf("Pos2 %i, %i, %i\n", tmp.x() != p.pos.x(), tmp.y() != p.pos.y(), tmp.z() != p.pos.z());
	// Assert that the position is correct!!
	assert(p.pos.x >= 0.0f && p.pos.x <= 1.0f &&
		p.pos.y >= 0.0f && p.pos.y <= 1.0f &&
		p.pos.z >= 0.0f && p.pos.z <= 1.0f);

	// ----------  DEFORMATION GRADIENT UPDATE ---------- //
	mat3 Ftmp = mm::mul(p.C, dt);
	mm::add_to(&Ftmp, 1.0f);
	mat3 F = mm::mul(Ftmp, p.F);

	const S3::property& p_prop = d_properties[p.prop_id];
	if (p_prop.plasticity)
	{
		mat3 u, v;
		vec3 s;

		svd(F.m[0][0], F.m[0][1], F.m[0][2],
			F.m[1][0], F.m[1][1], F.m[1][2],
			F.m[2][0], F.m[2][1], F.m[2][2],
			u.m[0][0], u.m[0][1], u.m[0][2],
			u.m[1][0], u.m[1][1], u.m[1][2],
			u.m[2][0], u.m[2][1], u.m[2][2],
			s.x, s.y, s.z,
			v.m[0][0], v.m[0][1], v.m[0][2],
			v.m[1][0], v.m[1][1], v.m[1][2],
			v.m[2][0], v.m[2][1], v.m[2][2]);

		// Snow paper elasticiy constrains
		s.x = glm::clamp(s.x, 1.0f - p_prop.t_c, 1.0f + p_prop.t_s);
		s.y = glm::clamp(s.y, 1.0f - p_prop.t_c, 1.0f + p_prop.t_s);
		s.z = glm::clamp(s.z, 1.0f - p_prop.t_c, 1.0f + p_prop.t_s);

		mat3 S;
		S.m[0][0] = s.x;
		S.m[1][1] = s.y;
		S.m[2][2] = s.z;

		const float oldJ = mm::determinant(F);

		F = mm::mul(u, mm::mul_trans(S, v));

		const float det = mm::determinant(F);
		const float newJ = glm::clamp(p.Jp * oldJ / det, p_prop.p_c, p_prop.p_s);

		p.Jp = newJ;
	}

	/*
	// ----------  PLASTICITY ---------- //
	// SVD of new deformation gradient F
	Eigen::JacobiSVD<Eigen::Matrix3f, Eigen::NoQRPreconditioner> svd(F, Eigen::ComputeFullU | Eigen::ComputeFullV);

	const S3::property& prop = d_properties[p.prop_id];
	if (prop.plasticity) {
		const Eigen::Matrix3f& svd_u = svd.matrixU();
		const Eigen::Matrix3f& svd_v = svd.matrixV();

		Eigen::Vector3f svd_e = svd.singularValues();

		// Snow paper elasticiy constrains
		for (int i = 0; i < 3; ++i) {
			svd_e[i] = glm::clamp(svd_e[i], 1.0f - prop.t_c, 1.0f + prop.t_s);
		}

		const float oldJ = F.determinant();
		F = svd_u * svd_e.asDiagonal() * svd_v.transpose();

		const float det = F.determinant();
		const float newJ = glm::clamp(p.Jp * oldJ / det, prop.p_c, prop.p_s);
		p.Jp = newJ;
	}*/

	p.F = F;

}
}

void loadParticles2GPU(const S3::Particle* particles, size_t num)
{
	static_assert(sizeof(S3::Particle) == sizeof(ParticleGPU),"Particle GPU must be equivalent to CPU one");
	static_assert(offsetof(S3::Particle, F) == offsetof(ParticleGPU, F), "Offsets not equivalent");
	static_assert(offsetof(S3::Particle, v) == offsetof(ParticleGPU, v), "Offsets not equivalent");
	static_assert(offsetof(S3::Particle, C) == offsetof(ParticleGPU, C), "Offsets not equivalent");
	static_assert(offsetof(S3::Particle, Jp) == offsetof(ParticleGPU, Jp), "Offsets not equivalent");


	ParticleGPU* tmp;
	gpuErrchk(hipMalloc((ParticleGPU**)&tmp, num * sizeof(ParticleGPU)));
	gpuErrchk(hipMemcpy(tmp, particles, num * sizeof(ParticleGPU), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_particles), (void**)&tmp, sizeof(ParticleGPU*)));

	h_numParticles = static_cast<unsigned int>(num);
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_numParticles), &h_numParticles, sizeof(unsigned int)));

}

void loadGrid2GPU(size_t num)
{
	assert(static_cast<size_t>(GRID_SIZE) == num);

	num = num * num * num;
	gpuErrchk(hipMalloc((vec4**)& h_grid, num * sizeof(vec4)));
	gpuErrchk(hipMemset(h_grid, 0, num * sizeof(vec4)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_grid), (void**)&h_grid, sizeof(vec4*)));
}

void loadPhysicsGrid2GPU(const Eigen::Array3f* grid, size_t num)
{
	assert(static_cast<size_t>(GRID_SIZE) == num);

	num = num * num * num;
	vec3* tmp;
	gpuErrchk(hipMalloc((vec3**) & tmp, num * sizeof(vec3)));
	gpuErrchk(hipMemcpy(tmp, grid, num * sizeof(vec3), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_physicsGrid), (void**)&tmp, sizeof(vec3*)));
}

void loadProperties2GPU(const S3::property* properties, size_t num)
{
	S3::property* tmp;
	gpuErrchk(hipMalloc((S3::property**) & tmp, num * sizeof(S3::property)));
	gpuErrchk(hipMemcpy(tmp, properties, num * sizeof(S3::property), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_properties), (void**)&tmp, sizeof(S3::property*)));
}

void storeParticles2Array(S3::Particle* particles, size_t num)
{
	S3::Particle* tmp;
	gpuErrchk(hipMemcpyFromSymbol(&tmp,HIP_SYMBOL(d_particles), sizeof(ParticleGPU*)));
	gpuErrchk(hipMemcpy(particles, tmp, num * sizeof(ParticleGPU), hipMemcpyDeviceToHost));
}

void clearGridGPU()
{
	size_t num = GRID_SIZE_I * GRID_SIZE_I * GRID_SIZE_I;
	gpuErrchk(hipMemset(h_grid, 0, num * sizeof(vec4)));
}

void runIterationGPU(float dt)
{
	clearGridGPU();
	gpuErrchk(hipDeviceSynchronize());// Coment this on release

	unsigned int numBlocks = (h_numParticles-1) / BLOCK_SIZE + 1;
	P2G_NeoHookean <<<numBlocks, BLOCK_SIZE >>> (dt);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());// Coment this on release


	int nBlocks = GRID_SIZE_I * GRID_SIZE_I * GRID_SIZE_I / BLOCK_SIZE;

	GridProcessing <<<nBlocks, BLOCK_SIZE >>> (dt);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());// Coment this on release

	G2P <<<numBlocks, BLOCK_SIZE >>> (dt);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());// Coment this on release
}